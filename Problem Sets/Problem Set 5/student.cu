#include "hip/hip_runtime.h"
/* Udacity HW5
   Histogramming for Speed

   The goal of this assignment is compute a histogram
   as fast as possible.  We have simplified the problem as much as
   possible to allow you to focus solely on the histogramming algorithm.

   The input values that you need to histogram are already the exact
   bins that need to be updated.  This is unlike in HW3 where you needed
   to compute the range of the data and then do:
   bin = (val - valMin) / valRange to determine the bin.

   Here the bin is just:
   bin = val

   so the serial histogram calculation looks like:
   for (i = 0; i < numElems; ++i)
     histo[val[i]]++;

   That's it!  Your job is to make it run as fast as possible!

   The values are normally distributed - you may take
   advantage of this fact in your implementation.

*/


#include "utils.h"

#define MAX_THREADS_PER_BLOCK 1024

__global__
void yourHisto(const unsigned int* const vals, //INPUT
               unsigned int* const histo,      //OUPUT
               int numVals)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= 0 && idx < numVals) {
    int bin = vals[idx];
    atomicAdd(histo + bin, 1);
  }
}

void computeHistogram(const unsigned int* const d_vals, //INPUT
                      unsigned int* const d_histo,      //OUTPUT
                      const unsigned int numBins,
                      const unsigned int numElems)
{
  const dim3 blockSize(MAX_THREADS_PER_BLOCK, 1, 1);
  const dim3 gridSize(numElems / blockSize.x + 1);

  yourHisto<<<gridSize, blockSize>>>(d_vals, d_histo, numElems);
}
